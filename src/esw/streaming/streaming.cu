#include <algorithm>
#include <cstdint>
#include <iostream>
#include <stdexcept>
#include <unordered_map>
#include <vector>

#include "streaming.hpp"

#include <hip/hip_runtime.h>
#include <nvEncodeAPI.h>

void NvCheck(NVENCSTATUS status) {
    if (status != NV_ENC_SUCCESS) {
        throw std::runtime_error("NvEnc failed");
    }
}

void cuCheck(hipError_t status) {
    if (status != hipSuccess) {
        throw std::runtime_error("CUDA failed");
    }
}

void cudaCheck(hipError_t status) {
    if (status != hipSuccess) {
        throw std::runtime_error("CUDA failed");
    }
}

namespace std {
    template<>
    struct equal_to<GUID> {
        bool operator()(GUID const& g1, GUID const& g2) const {
            return g1.Data1 == g2.Data1 && g1.Data2 == g2.Data2 && g1.Data3 == g2.Data3 &&
                   g1.Data4[0] == g2.Data4[0] && g1.Data4[1] == g2.Data4[1] && g1.Data4[2] == g2.Data4[2] &&
                   g1.Data4[3] == g2.Data4[3] && g1.Data4[4] == g2.Data4[4] && g1.Data4[5] == g2.Data4[5] &&
                   g1.Data4[6] == g2.Data4[6] && g1.Data4[7] == g2.Data4[7];
        }
    };

    template<>
    struct hash<GUID> {
        std::size_t operator()(GUID const& g) const {
            std::size_t seed = 0;
            seed ^= std::hash<std::uint32_t>{}(g.Data1);
            seed ^= std::hash<std::uint16_t>{}(g.Data2);
            seed ^= std::hash<std::uint16_t>{}(g.Data3);
            for (std::size_t i = 0; i < 8; ++i) {
                seed ^= std::hash<std::uint8_t>{}(g.Data4[i]);
            }
            return seed;
        }
    };
} // namespace std

std::unordered_map<GUID, std::string> GUID_TO_NAME{
        {NV_ENC_CODEC_HEVC_GUID, "HEVC"},
        {NV_ENC_CODEC_H264_GUID, "H264"},
        {NV_ENC_CODEC_AV1_GUID, "AV1"},
};

Streamer::Streamer(std::uint32_t width, std::uint32_t height) {
    cudaCheck(hipSetDevice(0));
    hipCtx_t context;
    cuCheck(hipCtxGetCurrent(&context));

    NvCheck(NvEncodeAPICreateInstance(&m_nvenc));
    NV_ENC_OPEN_ENCODE_SESSION_EX_PARAMS params{
            .version = NV_ENC_OPEN_ENCODE_SESSION_EX_PARAMS_VER,
            .deviceType = NV_ENC_DEVICE_TYPE_CUDA,
            .device = context,
            .apiVersion = NVENCAPI_VERSION,
    };
    void* encoder = nullptr;
    NvCheck(m_nvenc.nvEncOpenEncodeSessionEx(&params, &encoder));
    if (!encoder) {
        throw std::runtime_error("No encoder");
    }

    std::uint32_t guidCount;
    NvCheck(m_nvenc.nvEncGetEncodeGUIDCount(encoder, &guidCount));
    if (guidCount == 0) {
        throw std::runtime_error("No GUIDs");
    }

    std::vector<GUID> guids(guidCount);
    NvCheck(m_nvenc.nvEncGetEncodeGUIDs(encoder, guids.data(), guidCount, &guidCount));
    std::cout << "Supported encoders:" << std::endl;
    for (GUID const& guid: guids) {
        std::cout << "\t" << GUID_TO_NAME[guid] << std::endl;
    }

    GUID desiredEncodeGuid = NV_ENC_CODEC_HEVC_GUID;
    GUID desiredPresetGuid = NV_ENC_PRESET_P4_GUID;

    if (std::none_of(guids.begin(), guids.end(), [&](const GUID& guid) {
            return std::equal_to<GUID>{}(guid, desiredEncodeGuid);
        })) {
        throw std::runtime_error("No HEVC GUID");
    }

    std::uint32_t presetCount;
    NvCheck(m_nvenc.nvEncGetEncodePresetCount(encoder, desiredEncodeGuid, &presetCount));
    std::vector<GUID> presetGuids(presetCount);
    NvCheck(m_nvenc.nvEncGetEncodePresetGUIDs(encoder, desiredEncodeGuid, presetGuids.data(), presetCount, &presetCount));
    if (std::none_of(presetGuids.begin(), presetGuids.end(), [&](const GUID& guid) {
            return std::equal_to<GUID>{}(guid, desiredPresetGuid);
        })) {
        throw std::runtime_error("No P4 preset");
    }

    NV_ENC_TUNING_INFO tuningInfo = NV_ENC_TUNING_INFO_ULTRA_LOW_LATENCY;
    NV_ENC_PRESET_CONFIG presetConfig{
            .version = NV_ENC_PRESET_CONFIG_VER,
            .presetCfg = {
                    .version = NV_ENC_CONFIG_VER,
            },
    };
    NvCheck(m_nvenc.nvEncGetEncodePresetConfigEx(encoder, desiredEncodeGuid, desiredPresetGuid, tuningInfo, &presetConfig));

    NV_ENC_INITIALIZE_PARAMS encInitParams{
            .version = NV_ENC_INITIALIZE_PARAMS_VER,
            .encodeGUID = desiredEncodeGuid,
            .presetGUID = desiredPresetGuid,
            .encodeWidth = width,
            .encodeHeight = height,
            .darWidth = width,
            .darHeight = height,
            .frameRateNum = 30,
            .frameRateDen = 1,
            .tuningInfo = tuningInfo,
            .encodeConfig = &presetConfig.presetCfg,
    };
    NvCheck(m_nvenc.nvEncInitializeEncoder(encoder, &encInitParams));
}
