#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <NvOnnxParser.h>
#include <hip/hip_runtime_api.h>

#include "ioHelper.h"

#include <memory>
#include <string.h>

#include "inference.h"

#include <vector>

using namespace nvinfer1;


// struct Logger {

//     void log(nvinfer1::ILogger::Severity severity, nvinfer1::AsciiChar const* msg) {
//     }
// };


// static Logger logger;

// __global__ void forward() {
// }


// void setup() {
//     // TOOD: fix this
//     nvinfer1::createInferBuilder(logger);
// }

void InferenceNew::launchInference(IExecutionContext* context, hipStream_t stream, std::vector<float> const& inputTensor, std::vector<float>& outputTensor, void** bindings, int batchSize) {
    int inputId = InferenceNew::getBindingInputIndex(context);

    hipMemcpyAsync(bindings[inputId], inputTensor.data(), inputTensor.size() * sizeof(float), hipMemcpyHostToDevice, stream);

    context->enqueueV3(stream);

    hipMemcpyAsync(outputTensor.data(), bindings[1 - inputId], outputTensor.size() * sizeof(float), hipMemcpyDeviceToHost, stream);
}
